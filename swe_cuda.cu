#include "hip/hip_runtime.h"
//
// Created by luchin on 29-07-21.
//
#include <cassert>
#include <iostream>

static hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

Par_CUDA::Par_CUDA() : AbstractGoL() {
    int devId = 0;
    hipDeviceProp_t prop;
    checkCuda(hipGetDeviceProperties(&prop, devId));
    checkCuda(hipSetDevice(devId));
    int total_size = sizeof(char) * LARGO * LARGO;
    d_grid = nullptr;
    checkCuda(hipMalloc(&d_grid, total_size));
}

__global__ void step(char *grid) {
#ifdef CUDA_USE_2D
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
#else
    int tmp = blockIdx.x * blockDim.x + threadIdx.x;
    int x = tmp / LARGO;
    int y = tmp % LARGO;
#endif

    // contamos los vecinos
    //    printf("x is %d and y is %d\n", x, y);
    if (x > LARGO || y > LARGO) return;
    x += LARGO; // nos aseguramos de que x-1 sea positivo
    y += LARGO;
    int x_m = (x - 1) % LARGO;
    int x_p = (x + 1) % LARGO;
    int y_m = (y - 1) % LARGO;
    int y_p = (y + 1) % LARGO;
    x = x % LARGO;
    y = y % LARGO;
    int num_neighbors =
            grid[x_m * LARGO + y_m] + grid[x * LARGO + y_m] + grid[x_p * LARGO + y_m] +
            grid[x_m * LARGO + y] + grid[x_p * LARGO + y] +
            grid[x_m * LARGO + y_p] + grid[x * LARGO + y_p] + grid[x_p * LARGO + y_p];
    char alive = grid[x * LARGO + y];

    __syncthreads();
    // reemplazamos los lugares donde corresponde
    if ((alive && num_neighbors == 2) || num_neighbors == 3) {
        grid[x * LARGO + y] = 1;
    } else {
        grid[x * LARGO + y] = 0;
    }
}


void Par_CUDA::run_game(int num_steps) {
#ifdef CUDA_USE_2D
    dim3 dimGrid((LARGO + 7) / 8, (LARGO + 7) / 88, 1);
    dim3 dimBlock(8, 8, 1);
#else
    dim3 dimGrid((LARGO * LARGO + 7) / 8, 1, 1);
    dim3 dimBlock(8, 1, 1);
#endif
    hipMemcpy(d_grid, h_grid, sizeof(char) * LARGO * LARGO, hipMemcpyHostToDevice);
    for (int i = 0; i < num_steps; i++) {
        step<<<dimGrid, dimBlock>>>(d_grid);
    }
    hipMemcpy(h_grid, d_grid, sizeof(char) * LARGO * LARGO, hipMemcpyDeviceToHost);
}